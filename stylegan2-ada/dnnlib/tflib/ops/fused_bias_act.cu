#include "hip/hip_runtime.h"
// Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
//
// NVIDIA CORPORATION and its licensors retain all intellectual property
// and proprietary rights in and to this software, related documentation
// and any modifications thereto.  Any use, reproduction, disclosure or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA CORPORATION is strictly prohibited.

#define EIGEN_USE_GPU
#define __CUDA_INCLUDE_COMPILER_INTERNAL_HEADERS__
#include "tensorflow/core/framework/op.h"
#include "tensorflow/core/framework/op_kernel.h"
#include "tensorflow/core/framework/shape_inference.h"
#include <stdio.h>

using namespace tensorflow;
using namespace tensorflow::shape_inference;

#define OP_CHECK_CUDA_ERROR(CTX, CUDA_CALL) do { hipError_t err = CUDA_CALL; OP_REQUIRES(CTX, err == hipSuccess, errors::Internal(hipGetErrorName(err))); } while (false)

//------------------------------------------------------------------------
// CUDA kernel.

template <class T>
struct FusedBiasActKernelParams
{
    const T*    x;      // [sizeX]
    const T*    b;      // [sizeB] or NULL
    const T*    xref;   // [sizeX] or NULL
    const T*    yref;   // [sizeX] or NULL
    T*          y;      // [sizeX]

    int         grad;
    int         axis;
    int         act;
    float       alpha;
    float       gain;
    float       clamp;

    int         sizeX;
    int         sizeB;
    int         stepB;
    int         loopX;
};

template <class T>
static __global__ void FusedBiasActKernel(const FusedBiasActKernelParams<T> p)
{
    const float expRange        = 80.0f;
    const float halfExpRange    = 40.0f;
    const float seluScale       = 1.0507009873554804934193349852946f;
    const float seluAlpha       = 1.6732632423543772848170429916717f;

    // Loop over elements.
    int xi = blockIdx.x * p.loopX * blockDim.x + threadIdx.x;
    for (int loopIdx = 0; loopIdx < p.loopX && xi < p.sizeX; loopIdx++, xi += blockDim.x)
    {
        // Load and apply bias.
        float x = (float)p.x[xi];
        if (p.b)
            x += (float)p.b[(xi / p.stepB) % p.sizeB];
        float xref = (p.xref) ? (float)p.xref[xi] : 0.0f;
        float yref = (p.yref) ? (float)p.yref[xi] : 0.0f;
        float yy = (p.gain != 0.0f) ? yref / p.gain : 0.0f;

        // Evaluate activation func.
        float y;
        switch (p.act * 10 + p.grad)
        {
            // linear
            default:
            case 10: y = x; break;
            case 11: y = x; break;
            case 12: y = 0.0f; break;

            // relu
            case 20: y = (x > 0.0f) ? x : 0.0f; break;
            case 21: y = (yy > 0.0f) ? x : 0.0f; break;
            case 22: y = 0.0f; break;

            // lrelu
            case 30: y = (x > 0.0f) ? x : x * p.alpha; break;
            case 31: y = (yy > 0.0f) ? x : x * p.alpha; break;
            case 32: y = 0.0f; break;

            // tanh
            case 40: { float c = expf(x); float d = 1.0f / c; y = (x < -expRange) ? -1.0f : (x > expRange) ? 1.0f : (c - d) / (c + d); } break;
            case 41: y = x * (1.0f - yy * yy); break;
            case 42: y = x * (1.0f - yy * yy) * (-2.0f * yy); break;

            // sigmoid
            case 50: y = (x < -expRange) ? 0.0f : 1.0f / (expf(-x) + 1.0f); break;
            case 51: y = x * yy * (1.0f - yy); break;
            case 52: y = x * yy * (1.0f - yy) * (1.0f - 2.0f * yy); break;

            // elu
            case 60: y = (x >= 0.0f) ? x : expf(x) - 1.0f; break;
            case 61: y = (yy >= 0.0f) ? x : x * (yy + 1.0f); break;
            case 62: y = (yy >= 0.0f) ? 0.0f : x * (yy + 1.0f); break;

            // selu
            case 70: y = (x >= 0.0f) ? seluScale * x : (seluScale * seluAlpha) * (expf(x) - 1.0f); break;
            case 71: y = (yy >= 0.0f) ? x * seluScale : x * (yy + seluScale * seluAlpha); break;
            case 72: y = (yy >= 0.0f) ? 0.0f : x * (yy + seluScale * seluAlpha); break;

            // softplus
            case 80: y = (x > expRange) ? x : logf(expf(x) + 1.0f); break;
            case 81: y = x * (1.0f - expf(-yy)); break;
            case 82: { float c = expf(-yy); y = x * c * (1.0f - c); } break;

            // swish
            case 90: y = (x < -expRange) ? 0.0f : x / (expf(-x) + 1.0f); break;
            case 91:
            case 92:
                {
                    float c = expf(xref);
                    float d = c + 1.0f;
                    if (p.grad == 1)
                        y = (xref > halfExpRange) ? x : x * c * (xref + d) / (d * d);
                    else
                        y = (xref > halfExpRange) ? 0.0f : x * c * (xref * (2.0f - d) + 2.0f * d) / (d * d * d);
                    yref = (xref < -expRange) ? 0.0f : xref / (expf(-xref) + 1.0f) * p.gain;
                }
                break;
        }

        // Apply gain.
        y *= p.gain;

        // Clamp.
        if (p.clamp >= 0.0f)
        {
            if (p.grad == 0)
                y = (fabsf(y) < p.clamp) ? y : (y >= 0.0f) ? p.clamp : -p.clamp;
            else
                y = (fabsf(yref) < p.clamp) ? y : 0.0f;
        }

        // Store.
        p.y[xi] = (T)y;
    }
}

//------------------------------------------------------------------------
// TensorFlow op.

template <class T>
struct FusedBiasActOp : public OpKernel
{
    FusedBiasActKernelParams<T> m_attribs;

    FusedBiasActOp(OpKernelConstruction* ctx) : OpKernel(ctx)
    {
        memset(&m_attribs, 0, sizeof(m_attribs));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("grad",    &m_attribs.grad));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("axis",    &m_attribs.axis));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("act",     &m_attribs.act));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("alpha",   &m_attribs.alpha));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("gain",    &m_attribs.gain));
        OP_REQUIRES_OK(ctx, ctx->GetAttr("clamp",   &m_attribs.clamp));
        OP_REQUIRES(ctx, m_attribs.grad >= 0, errors::InvalidArgument("grad must be non-negative"));
        OP_REQUIRES(ctx, m_attribs.axis >= 0, errors::InvalidArgument("axis must be non-negative"));
        OP_REQUIRES(ctx, m_attribs.act >= 0, errors::InvalidArgument("act must be non-negative"));
    }

    void Compute(OpKernelContext* ctx)
    {
        FusedBiasActKernelParams<T> p = m_attribs;
        hipStream_t stream = ctx->eigen_device<Eigen::GpuDevice>().stream();

        const Tensor& x     = ctx->input(0); // [...]
        const Tensor& b     = ctx->input(1); // [sizeB] or [0]
        const Tensor& xref  = ctx->input(2); // x.shape or [0]
        const Tensor& yref  = ctx->input(3); // x.shape or [0]
        p.x = x.flat<T>().data();
        p.b = (b.NumElements()) ? b.flat<T>().data() : NULL;
        p.xref = (xref.NumElements()) ? xref.flat<T>().data() : NULL;
        p.yref = (yref.NumElements()) ? yref.flat<T>().data() : NULL;
        OP_REQUIRES(ctx, b.NumElements() == 0 || m_attribs.axis < x.dims(), errors::InvalidArgument("axis out of bounds"));
        OP_REQUIRES(ctx, b.dims() == 1, errors::InvalidArgument("b must have rank 1"));
        OP_REQUIRES(ctx, b.NumElements() == 0 || b.NumElements() == x.dim_size(m_attribs.axis), errors::InvalidArgument("b has wrong number of elements"));
        OP_REQUIRES(ctx, xref.NumElements() == 0 || xref.NumElements() == x.NumElements(), errors::InvalidArgument("xref has wrong number of elements"));
        OP_REQUIRES(ctx, yref.NumElements() == 0 || yref.NumElements() == x.NumElements(), errors::InvalidArgument("yref has wrong number of elements"));
        OP_REQUIRES(ctx, x.NumElements() <= kint32max, errors::InvalidArgument("x is too large"));

        p.sizeX = (int)x.NumElements();
        p.sizeB = (int)b.NumElements();
        p.stepB = 1;
        for (int i = m_attribs.axis + 1; i < x.dims(); i++)
            p.stepB *= (int)x.dim_size(i);

        Tensor* y = NULL; // x.shape
        OP_REQUIRES_OK(ctx, ctx->allocate_output(0, x.shape(), &y));
        p.y = y->flat<T>().data();

        p.loopX = 4;
        int blockSize = 4 * 32;
        int gridSize = (p.sizeX - 1) / (p.loopX * blockSize) + 1;
        void* args[] = {&p};
        OP_CHECK_CUDA_ERROR(ctx, hipLaunchKernel((void*)FusedBiasActKernel<T>, gridSize, blockSize, args, 0, stream));
    }
};

REGISTER_OP("FusedBiasAct")
    .Input      ("x: T")
    .Input      ("b: T")
    .Input      ("xref: T")
    .Input      ("yref: T")
    .Output     ("y: T")
    .Attr       ("T: {float, half}")
    .Attr       ("grad: int = 0")
    .Attr       ("axis: int = 1")
    .Attr       ("act: int = 0")
    .Attr       ("alpha: float = 0.0")
    .Attr       ("gain: float = 1.0")
    .Attr       ("clamp: float = -1.0");
REGISTER_KERNEL_BUILDER(Name("FusedBiasAct").Device(DEVICE_GPU).TypeConstraint<float>("T"), FusedBiasActOp<float>);
REGISTER_KERNEL_BUILDER(Name("FusedBiasAct").Device(DEVICE_GPU).TypeConstraint<Eigen::half>("T"), FusedBiasActOp<Eigen::half>);

//------------------------------------------------------------------------
